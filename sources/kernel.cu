#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"
#include "Context.h"
#include <stdio.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ float2 LineClosestPoint(float2 a, float2 b, float2 p)
{
	float2 ap = p - a;
	float2 ab_dir = b - a;
	float dot = ap.x * ab_dir.x + ap.y * ab_dir.y;
	if (dot < 0.0f)
		return a;
	float ab_len_sqr = ab_dir.x * ab_dir.x + ab_dir.y * ab_dir.y;
	if (dot > ab_len_sqr)
		return b;
	return a + ab_dir * dot / ab_len_sqr;
}

__global__ void kernel(float* A, float* v, int stride) 
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;

	float2 p;
	p.x = float(j) * 256.0f / 224;// / gridDim.x / blockDim.x;
	p.y = float(i) * 256.0f / 224;// / gridDim.y / blockDim.y;
		
	int it = 0;
	int offsetx = stride * 2 * k;
	int offsety = stride * (2 * k + 1);

	float min_dist = 1e6f;
	for (;it < stride; ++it)
	{
		float2 p0;
		p0.x = v[offsetx + it];
		p0.y = v[offsety + it];
		float2 p1;
		p1.x = v[offsetx + it + 1];
		p1.y = v[offsety + it + 1];

		if (p1.x == p0.x && p0.y == p1.y)
		{
			it += 1;
			continue;
		}
		if (p1.x <0 && p1.y <0)
		{
			break;
		}

		float2 x = LineClosestPoint(p0, p1, p);
		float2 d = x - p;
		float distance = hypotf(d.x, d.y);
		if (min_dist > distance)
		{
			min_dist = distance;
		}
	}

	A[j + i * gridDim.x * blockDim.x + k * gridDim.x * blockDim.x * gridDim.y * blockDim.y] = expf(-min_dist * min_dist / 2.0f);
}


DeviceGuard::DeviceGuard(int device)
{
	gpuErrchk(hipGetDevice(&backup_device));
	gpuErrchk(hipSetDevice(device));
}

DeviceGuard::~DeviceGuard()
{
	gpuErrchk(hipSetDevice(backup_device));
}

Context InitContext(int stride, int size_x, int size_y, int size_z, int device)
{
	Context ctx;
	ctx.device = device;
	DeviceGuard cuda(ctx.device);

	gpuErrchk(hipMalloc(&ctx.vector_array, 2 * stride * size_z * sizeof(float)));
	ctx.size_x = size_x;
	ctx.size_y = size_y;
	ctx.size_z = size_z;
	ctx.stride = stride;

	gpuErrchk(hipMalloc(&ctx.raster_array, size_z * size_x * size_y * sizeof(float)));
	ctx.raster_host = new float[size_z * size_x * size_y];
	ctx.vector_host = new float[2 * stride * size_z];

	return ctx;
}

void FreeContext(Context* ctx)
{
	DeviceGuard cuda(ctx->device);

	gpuErrchk(hipFree(ctx->vector_array));
	gpuErrchk(hipFree(ctx->raster_array));
	delete[] ctx->raster_host;
	delete[] ctx->vector_host;
}

void Render(Context* ctx, int count)
{
	DeviceGuard cuda(ctx->device);

	int threads_in_block_x = 8;
	int threads_in_block_y = 8;

	int blocks_x = ctx->size_x / threads_in_block_x;
	int blocks_y = ctx->size_y / threads_in_block_y;

	gpuErrchk(hipMemcpy(ctx->vector_array, ctx->vector_host, 2 * ctx->stride * count * sizeof(float), hipMemcpyHostToDevice));

	kernel<<<dim3(blocks_x, blocks_y, count), dim3(threads_in_block_x, threads_in_block_y, 1)>>>(ctx->raster_array, ctx->vector_array, ctx->stride);

	gpuErrchk(hipPeekAtLastError());

	gpuErrchk(hipMemcpy(ctx->raster_host, ctx->raster_array, count * ctx->size_x * ctx->size_y * sizeof(float), hipMemcpyDeviceToHost));
}
